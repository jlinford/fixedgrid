#include "hip/hip_runtime.h"
/*
 *  discretize.c
 *  
 *  Transport module.
 *  Main kernel.
 *
 *  Created by John Linford on 4/8/08.
 *  Copyright 2008 Transatlantic Giraffe. All rights reserved.
 *
 */

#include "discretize.h"
#include "timer.h"
#include "common.h"
#include "params.h"
#include "cuda_grid.h"

typedef struct neighbors
{
    real_t left[2];
    real_t right[2];
} neighbors_t;

/**
 * Upwinded advection/diffusion equation
 */
extern "C" __device__
real_t advec_diff(real_t cell_size,
                  real_t c2l, real_t w2l, real_t d2l, 
                  real_t c1l, real_t w1l, real_t d1l, 
                  real_t   c, real_t   w, real_t   d, 
                  real_t c1r, real_t w1r, real_t d1r, 
                  real_t c2r, real_t w2r, real_t d2r)
{
    real_t wind, diff_term, advec_term;
    real_t advec_termR, advec_termL;

    wind = (w1l + w) / 2.0;
    if(wind >= 0.0) advec_termL = (1.0/6.0) * ( -c2l + 5.0*c1l + 2.0*c );
    else advec_termL = (1.0/6.0) * ( 2.0*c1l + 5.0*c - c1r );
    advec_termL *= wind;
    wind = (w1r + w) / 2.0;
    if(wind >= 0.0) advec_termR = (1.0/6.0) * ( -c1l + 5.0*c + 2.0*c1r );
    else advec_termR = (1.0/6.0) * ( 2.0*c + 5.0*c1r - c2r );
    advec_termR *= wind;
    advec_term = (advec_termL - advec_termR) / cell_size;
    diff_term = ( ((d1l+d)/2)*(c1l-c) - ((d+d1r)/2)*(c-c1r) ) / (cell_size * cell_size);
    return advec_term + diff_term;
}

/**
 * Collects neighborhood data into simple struct
 */
extern "C" __device__
void get_x_neighbors(neighbors_t* n, real_t field[NZ][NY][NX])
{
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    if(blockIdx.x == 0 && threadIdx.x == 0)
    {
        n->left[0] = field[z][y][NX-2];
        n->left[1] = field[z][y][NX-1];
    }
    if(blockIdx.x == 0 && threadIdx.x == 1)
    {
        n->left[0] = field[z][y][NX-1];
        n->left[1] = field[z][y][0];
    }
    if(blockIdx.x > 0 || threadIdx.x > 1)
    {
        n->left[0] = field[z][y][x-2];
        n->left[1] = field[z][y][x-1];
    }
    
    if(blockIdx.x == gridDim.x-1 && threadIdx.x == BLOCK_X-1)
    {
        n->right[0] = field[z][y][1];
        n->right[1] = field[z][y][0];
    }
    if(blockIdx.x == gridDim.x-1 && threadIdx.x == BLOCK_X-2)
    {
        n->right[0] = field[z][y][0];
        n->right[1] = field[z][y][NX-1];
    }
    if(blockIdx.x < gridDim.x-1 || threadIdx.x < BLOCK_X-2)
    {
        n->right[0] = field[z][y][x+2];
        n->right[1] = field[z][y][x+1];
    }
}

/**
 * Collects neighborhood data into simple struct
 */
extern "C" __device__
void get_y_neighbors(neighbors_t* n, real_t field[NZ][NY][NX])
{
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    if(blockIdx.y == 0 && threadIdx.y == 0)
    {
        n->left[0] = field[z][NY-2][x];
        n->left[1] = field[z][NY-1][x];
    }
    if(blockIdx.y == 0 && threadIdx.y == 1)
    {
        n->left[0] = field[z][NY-1][x];
        n->left[1] = field[z][0][x];
    }
    if(blockIdx.y > 0 || threadIdx.y > 1)
    {
        n->left[0] = field[z][y-2][x];
        n->left[1] = field[z][y-1][x];
    }
    
    if(blockIdx.y == gridDim.y-1 && threadIdx.y == BLOCK_Y-1)
    {
        n->right[0] = field[z][1][x];
        n->right[1] = field[z][0][x];
    }
    if(blockIdx.y == gridDim.y-1 && threadIdx.y == BLOCK_Y-2)
    {
        n->right[0] = field[z][0][x];
        n->right[1] = field[z][NX-1][x];
    }
    if(blockIdx.y < gridDim.y-1 || threadIdx.y < BLOCK_Y-2)
    {
        n->right[0] = field[z][y+2][x];
        n->right[1] = field[z][y+1][x];
    }
}

/**
 * Collects neighborhood data into simple struct
 */
extern "C" __device__
void get_z_neighbors(neighbors_t* n, real_t field[NZ][NY][NX])
{
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    if(blockIdx.z == 0 && threadIdx.z == 0)
    {
        n->left[0] = field[NZ-2][y][x];
        n->left[1] = field[NZ-1][y][x];
    }
    if(blockIdx.z == 0 && threadIdx.z == 1)
    {
        n->left[0] = field[NZ-1][y][x];
        n->left[1] = field[0][y][x];
    }
    if(blockIdx.z > 0 || threadIdx.z > 1)
    {
        n->left[0] = field[z-2][y][x];
        n->left[1] = field[z-1][y][x];
    }
    
    if(blockIdx.z == gridDim.z-1 && threadIdx.z == BLOCK_Z-1)
    {
        n->right[0] = field[1][y][x];
        n->right[1] = field[0][y][x];
    }
    if(blockIdx.z == gridDim.z-1 && threadIdx.z == BLOCK_Z-2)
    {
        n->right[0] = field[0][y][x];
        n->right[1] = field[NZ-1][y][x];
    }
    if(blockIdx.z < gridDim.z-1 || threadIdx.z < BLOCK_Z-2)
    {
        n->right[0] = field[z+2][y][x];
        n->right[1] = field[z+1][y][x];
    }
}

/**
 * Initialize discretization kernel data
 */
extern "C" __global__
void discretize_init(real_t c_in[NZ][NY][NX], 
                     real_t buff[NZ][NY][NX],
                     real_t c_out[NZ][NY][NX])
{
    // Data index
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    buff[z][y][x] = c_out[z][y][x] = c_in[z][y][x];
}

/**
 * Finalize discretization
 */
extern "C" __global__
void discretize_final(real_t c_in[NZ][NY][NX],
                      real_t buff[NZ][NY][NX], 
                      real_t c_out[NZ][NY][NX])
{
    // Data index
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;

    // Average results into c_out
    c_out[z][y][x] = 0.5 * (c_out[z][y][x] + buff[z][y][x]);
    if(c_out[z][y][x] < 0.0) c_out[z][y][x] = 0.0;
    
    // Update original conc data for next discretization
    c_in[z][y][x] = c_out[z][y][x];
}

/**
 * X-discretization
 */
extern "C" __global__
void advec_diff_x(real_t cell_size, real_t dt,
                  real_t c_in[NZ][NY][NX], 
                  real_t wind[NZ][NY][NX], 
                  real_t diff[NZ][NY][NX], 
                  real_t buff[NZ][NY][NX])
{
    // Data index
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    // Change in conc
    real_t dcdx;
    
    // Data
    neighbors_t conc_n;
    neighbors_t wind_n;
    neighbors_t diff_n;
    
    // Prepare for discretization
    get_x_neighbors(&conc_n, c_in);
    get_x_neighbors(&wind_n, wind);
    get_x_neighbors(&diff_n, diff);
        
    // Discretize
    dcdx = advec_diff(cell_size, 
                      conc_n.left[0], wind_n.left[0], diff_n.left[0],
                      conc_n.left[1], wind_n.left[1], diff_n.left[1],
                       c_in[z][y][x],  wind[z][y][x],  diff[z][y][x],
                      conc_n.right[1], wind_n.right[1], diff_n.right[1],
                      conc_n.right[0], wind_n.right[0], diff_n.right[0]);
    buff[z][y][x] += dt*dcdx;
}

/**
 * X-discretization
 */
extern "C" __global__
void advec_diff_y(real_t cell_size, real_t dt,
                  real_t c_in[NZ][NY][NX], 
                  real_t wind[NZ][NY][NX], 
                  real_t diff[NZ][NY][NX], 
                  real_t buff[NZ][NY][NX])
{
    // Data index
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    // Change in conc
    real_t dcdx;
    
    // Data
    neighbors_t conc_n;
    neighbors_t wind_n;
    neighbors_t diff_n;
    
    // Prepare for discretization
    get_y_neighbors(&conc_n, c_in);
    get_y_neighbors(&wind_n, wind);
    get_y_neighbors(&diff_n, diff);
        
    // Discretize
    dcdx = advec_diff(cell_size, 
                      conc_n.left[0], wind_n.left[0], diff_n.left[0],
                      conc_n.left[1], wind_n.left[1], diff_n.left[1],
                       c_in[z][y][x],  wind[z][y][x],  diff[z][y][x],
                      conc_n.right[1], wind_n.right[1], diff_n.right[1],
                      conc_n.right[0], wind_n.right[0], diff_n.right[0]);
    buff[z][y][x] += dt*dcdx;
}

/**
 * X-discretization
 */
extern "C" __global__
void advec_diff_z(real_t cell_size, real_t dt,
                  real_t c_in[NZ][NY][NX], 
                  real_t wind[NZ][NY][NX], 
                  real_t diff[NZ][NY][NX], 
                  real_t buff[NZ][NY][NX])
{
    // Data index
    int x = blockIdx.x*BLOCK_X + threadIdx.x;
    int y = blockIdx.y*BLOCK_Y + threadIdx.y;
    int z = blockIdx.z*BLOCK_Z + threadIdx.z;
    
    // Change in conc
    real_t dcdx;
    
    // Data
    neighbors_t conc_n;
    neighbors_t wind_n;
    neighbors_t diff_n;
    
    // Prepare for discretization
    get_z_neighbors(&conc_n, c_in);
    get_z_neighbors(&wind_n, wind);
    get_z_neighbors(&diff_n, diff);
        
    // Discretize
    dcdx = advec_diff(cell_size, 
                      conc_n.left[0], wind_n.left[0], diff_n.left[0],
                      conc_n.left[1], wind_n.left[1], diff_n.left[1],
                       c_in[z][y][x],  wind[z][y][x],  diff[z][y][x],
                      conc_n.right[1], wind_n.right[1], diff_n.right[1],
                      conc_n.right[0], wind_n.right[0], diff_n.right[0]);
    buff[z][y][x] += dt*dcdx;
}